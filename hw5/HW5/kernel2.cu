#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int resX, int resY, int *img, int maxIterations, int pitch)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int thisX = blockIdx.x * blockDim.x + threadIdx.x; // for i
    int thisY = blockIdx.y * blockDim.y + threadIdx.y; // for j

    if (thisX >= resX) //unuse position by pitching
        return;

    float x = lowerX + thisX * stepX; //  float x = x0 + i * dx;
    float y = lowerY + thisY * stepY; //  float y = y0 + j * dy;

    int index = thisY * pitch + thisX;
    img[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    //__host__ for cpu call hostFE(x1, y1, x0, y0, output, width, height, maxIterations);
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    size_t pitch_in_byte;
    int *host_res;
    int *kernel_res;

    hipMallocPitch((void **)&kernel_res, &pitch_in_byte, sizeof(int) * resX, resY);
    hipHostAlloc(&host_res, resY * pitch_in_byte, hipHostMallocMapped);

    int pitch = pitch_in_byte / sizeof(int);

    dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numblock(resX / threadPerBlock.x, resY / threadPerBlock.y);
    // cudaHostGetDevicePointer( &kernel_res, host_res, 0 );

    mandelKernel<<<numblock, threadPerBlock>>>(lowerX, lowerY, stepX, stepY, resX, resY, kernel_res, maxIterations, pitch);
    
    hipMemcpy(host_res, kernel_res, resY * pitch_in_byte, hipMemcpyDeviceToHost);
    for (int j = 0; j < resY; j++)
    {
        for (int i = 0; i < resX; i++)
        {
            img[j * resX + i] = host_res[j * pitch + i];
        }
    }
    hipHostFree(host_res);
    hipFree(kernel_res);
}
