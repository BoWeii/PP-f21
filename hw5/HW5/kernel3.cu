#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define PIXEL_PER_THREAD 2

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int resX, int resY, int *img, int maxIterations, int pitch)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * PIXEL_PER_THREAD; // for i
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * PIXEL_PER_THREAD; // for j

    for (int j = thisY; j < PIXEL_PER_THREAD + thisY; j++)
    {
        if (j >= resY) //unuse position by pitching
            break;
        for (int i = thisX; i < PIXEL_PER_THREAD + thisX; i++)
        {
            if (i >= resX) //unuse position by pitching
                break;

            float x = lowerX + i * stepX; //  float x = x0 + i * dx;
            float y = lowerY + j * stepY; //  float y = y0 + j * dy;

            int index = j * pitch + i;
            img[index] = mandel(x, y, maxIterations);
        }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    //__host__ for cpu call hostFE(x1, y1, x0, y0, output, width, height, maxIterations);
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    size_t pitch_in_byte;
    int *host_res;
    int *kernel_res;

    hipMallocPitch((void **)&kernel_res, &pitch_in_byte, sizeof(int) * resX, resY);
    hipHostAlloc(&host_res, resY * pitch_in_byte, hipHostMallocMapped);

    int pitch = pitch_in_byte / sizeof(int);

    int num_x = (int)ceil((float)resX / (BLOCK_SIZE * PIXEL_PER_THREAD));
    int num_y = (int)ceil((float)resY / (BLOCK_SIZE * PIXEL_PER_THREAD));
    dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numblock(num_x, num_y);
    // cudaHostGetDevicePointer( &kernel_res, host_res, 0 );

    mandelKernel<<<numblock, threadPerBlock>>>(lowerX, lowerY, stepX, stepY, resX, resY, kernel_res, maxIterations, pitch);
    hipDeviceSynchronize();
    hipMemcpy(host_res, kernel_res, resY * pitch_in_byte, hipMemcpyDeviceToHost);
    for (int j = 0; j < resY; j++)
    {
        for (int i = 0; i < resX; i++)
        {
            img[j * resX + i] = host_res[j * pitch + i];
        }
    }
    hipHostFree(host_res);
    hipFree(kernel_res);
}
